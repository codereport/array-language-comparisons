#include "hip/hip_runtime.h"
#include "matx.h"
#include <cassert>
#include <cstdio>
#include <math.h>
#include <memory>

using namespace matx;

/**
 * MatX uses C++ expression templates to build arithmetic expressions that compile into a lazily-evaluated
 * type for executing on the device. Currently, nvcc cannot see certain optimizations
 * when building the expression tree that would be obvious by looking at the code. Specifically any code reusing
 * the same tensor multiple times appears to the compiler as separate tensors, and it may issue multiple load
 * instructions. While caching helps, this can have a slight performance impact when compared to native CUDA
 * kernels. To work around this problem, complex expressions can be placed in a custom operator by adding some
 * boilerplate code around the original expression. This custom operator can then be used either alone or inside
 * other arithmetic expressions, and only a single load is issues for each tensor.
 *
 * This example uses the Black-Scholes equtation to demonstrate the two ways to implement the equation in MatX, and
 * shows the performance difference.
 */

/* Custom operator */
template <class O, class I1, class I2, class I3, class I4, class I5>
class BlackScholes : public BaseOp<BlackScholes<O, I1, I2, I3, I4, I5>>
{
private:
    O out_;
    I1 K_;
    I2 V_;
    I3 S_;
    I4 r_;
    I5 T_;

public:
    BlackScholes(O out, I1 K, I2 V, I3 S, I4 r, I5 T)
        : out_(out), V_(V), S_(S), K_(K), r_(r), T_(T) {}

    __device__ inline void operator()(index_t idx)
    {
        auto V = V_();
        auto K = K_();
        auto S = S_(idx);
        auto T = T_();
        auto r = r_();

        auto VsqrtT = V * sqrt(T);
        auto d1 = (log(S / K) + (r + 0.5f * V * V) * T) / VsqrtT;
        auto d2 = d1 - VsqrtT;
        auto cdf_d1 = normcdff(d1);
        auto cdf_d2 = normcdff(d2);
        auto expRT = exp(-1.f * r * T);

        out_(idx) = S * cdf_d1 - K * expRT * cdf_d2;
    }

    __host__ __device__ inline index_t Size(uint32_t i) const { return out_.Size(i); }
    static inline constexpr __host__ __device__ int32_t Rank() { return O::Rank(); }
};

template <typename T1>
void compute_black_scholes_matx(tensor_t<T1, 0> &K,
                                tensor_t<T1, 1> &S,
                                tensor_t<T1, 0> &V,
                                tensor_t<T1, 0> &r,
                                tensor_t<T1, 0> &T,
                                tensor_t<T1, 1> &output,
                                cudaExecutor &exec)
{
    auto VsqrtT = V * sqrt(T);
    auto d1 = (log(S / K) + (r + 0.5f * V * V) * T) / VsqrtT;
    auto d2 = d1 - VsqrtT;
    auto cdf_d1 = normcdf(d1);
    auto cdf_d2 = normcdf(d2);
    auto expRT = exp(-1.f * r * T);

    (output = S * cdf_d1 - K * expRT * cdf_d2).run(exec);
}

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
    MATX_ENTER_HANDLER();

    using dtype = float;

    index_t input_size = 100000;
    auto output_tensor = make_tensor<dtype>({input_size});
    auto S_tensor = make_tensor<dtype>({input_size});
    auto K_tensor = make_tensor<dtype>({});
    auto V_tensor = make_tensor<dtype>({});
    auto r_tensor = make_tensor<dtype>({});
    auto T_tensor = make_tensor<dtype>({});
    float time_ms;
    int num_iterations = 99;

    for (index_t i = 0; i < input_size; i++)
    {
        S_tensor(i) = (dtype)90 + dtype(i % 20);
    }
    K_tensor() = (dtype)100.;
    V_tensor() = (dtype)0.1;
    r_tensor() = (dtype)0.05;
    T_tensor() = (dtype)1.0;

    hipStream_t stream;
    hipStreamCreate(&stream);
    cudaExecutor exec{stream};
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    BlackScholes(output_tensor, K_tensor, V_tensor, S_tensor, r_tensor, T_tensor).run(exec);
    exec.sync();

    hipEventRecord(start, stream);
    for (int i = 0; i < num_iterations; i++)
    {
        BlackScholes(output_tensor, K_tensor, V_tensor, S_tensor, r_tensor, T_tensor).run(exec);
    }
    hipEventRecord(stop, stream);
    exec.sync();
    hipEventElapsedTime(&time_ms, start, stop);

    printf("Black-Scholes time = %.2fus per iteration\n",
           time_ms * 1e3 / num_iterations);

    compute_black_scholes_matx(K_tensor, S_tensor, V_tensor, r_tensor, T_tensor, output_tensor, exec);

    printf("First 20 values of computed Black-Scholes output:\n");
    for (index_t i = 0; i < 20; i++)
    {
        printf("%f\n", static_cast<float>(output_tensor(i)));
    }

    hipStreamDestroy(stream);
    MATX_CUDA_CHECK_LAST_ERROR();
    MATX_EXIT_HANDLER();
}
