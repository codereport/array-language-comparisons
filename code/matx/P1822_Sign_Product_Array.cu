#include "hip/hip_runtime.h"

#include <matx.h>

auto array_sign(matx::tensor_t<int32_t, 1> nums) -> matx::tensor_t<int32_t, 0> {
  auto sign = matx::make_tensor<int32_t>({});
  (sign = matx::prod(matx::max(matx::min(nums, 1), -1))).run();
  return sign;
}

auto main() -> int {

  auto t = matx::make_tensor<int32_t>({7});
  t.SetVals({-1, -2, -3, -4, 3, 2, 1});
  matx::print(array_sign(t));

  auto t2 = matx::make_tensor<int32_t>({5});
  t2.SetVals({1, 5, 0, 2, -3});
  matx::print(array_sign(t2));

  return 0;
}
