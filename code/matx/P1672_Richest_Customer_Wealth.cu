#include "hip/hip_runtime.h"

#include <matx.h>

auto max_wealth(matx::tensor_t<int32_t, 2> accounts) -> matx::tensor_t<int32_t, 0> {
  auto max = matx::make_tensor<int32_t>({});
  (max = matx::rmax(matx::sum(accounts, {0}))).run();
  return max;
}

auto main() -> int {

  auto t = matx::make_tensor<int32_t>({2, 3});
  t.SetVals({{1, 2, 3}, {3, 2, 1}});
  matx::print(max_wealth(t)); // 6

  auto t2 = matx::make_tensor<int32_t>({3, 2});
  t2.SetVals({{1, 5}, {7, 3}, {3, 5}});
  matx::print(max_wealth(t2)); // 10

  auto t3 = matx::make_tensor<int32_t>({3, 3});
  t3.SetVals({{2, 8, 7}, {7, 1, 3}, {1, 9, 5}});
  matx::print(max_wealth(t3)); // 17

  return 0;
}
